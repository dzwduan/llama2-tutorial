#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "../solve.h"

int test_relu_example1() {
    printf("  Test 1:\n");
    
    int size = 5;
    float h_input[] = {-2.0f, -1.0f, 0.0f, 1.0f, 2.0f};
    float h_output[5] = {0};
    float expected[] = {0.0f, 0.0f, 0.0f, 1.0f, 2.0f};
    
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));
    
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    
    solve_relu(d_input, d_output, size);
    
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("    Input:    [%.1f, %.1f, %.1f, %.1f, %.1f]\n", 
           h_input[0], h_input[1], h_input[2], h_input[3], h_input[4]);
    printf("    Output:   [%.1f, %.1f, %.1f, %.1f, %.1f]\n", 
           h_output[0], h_output[1], h_output[2], h_output[3], h_output[4]);
    printf("    Expected: [%.1f, %.1f, %.1f, %.1f, %.1f]\n", 
           expected[0], expected[1], expected[2], expected[3], expected[4]);
    
    int passed = 1;
    for (int i = 0; i < size; i++) {
        if (fabsf(h_output[i] - expected[i]) > 1e-6f) {
            passed = 0;
            break;
        }
    }
    
    hipFree(d_input); hipFree(d_output);
    
    printf("    %s\n", passed ? "PASS" : "FAIL");
    return passed;
}

int test_relu_example2() {
    printf("  Test 2:\n");
    
    int size = 3;
    float h_input[] = {-3.5f, 0.0f, 4.2f};
    float h_output[3] = {0};
    float expected[] = {0.0f, 0.0f, 4.2f};
    
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));
    
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    
    solve_relu(d_input, d_output, size);
    
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("    Input:    [%.1f, %.1f, %.1f]\n", h_input[0], h_input[1], h_input[2]);
    printf("    Output:   [%.1f, %.1f, %.1f]\n", h_output[0], h_output[1], h_output[2]);
    printf("    Expected: [%.1f, %.1f, %.1f]\n", expected[0], expected[1], expected[2]);
    
    int passed = 1;
    for (int i = 0; i < size; i++) {
        if (fabsf(h_output[i] - expected[i]) > 1e-6f) {
            passed = 0;
            break;
        }
    }
    
    hipFree(d_input); hipFree(d_output);
    
    printf("    %s\n", passed ? "PASS" : "FAIL");
    return passed;
}
// Conditional compilation: only include main function when testing standalone
#ifdef STANDALONE_TEST
int main() {
    printf("CUDA ReLU Activation Tests\n");
    printf("--------------------------\n");
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("ERROR: No CUDA devices found\n");
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU: %s\n\n", prop.name);
    
    printf("Running tests...\n");
    
    int passed = 0;
    if (test_relu_example1()) passed++;
    if (test_relu_example2()) passed++;
    
    printf("\nResults: %d/3 tests passed\n", passed);
    
    if (passed == 3) {
        printf("All tests passed\n");
        return 0;
    } else {
        printf("Some tests failed\n");
        return 1;
    }
}
#endif